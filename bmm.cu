#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 

#define TILEX 32
#define TILEY 16

#if (TILEX > TILEY)
#define TILEMAX TILEX
#define TILEMIN TILEY
#else
#define TILEMAX TILEY
#define TILEMIN TILEX
#endif // (TILEX > TILEY)


// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n)
{

	// write your GPU kernel function here
	__shared__ float As[TILEY][TILEMAX]; // = A[bx*TILEY:(bx+1)*TILEY][:]
	__shared__ float Bs[TILEMAX][TILEX]; // = B[:][bx*TILEX:(bx+1)*TILEX]

	float temp = 0;
	for (int index = 0; index < (n/TILEMAX); index++)
	{
		for (int asIndex = 0; asIndex < (TILEMAX / TILEX); asIndex++)
		{
			As[ty][TILEX * asIndex + tx] = mem2d(ad, m, (by * TILEY + ty), (index * TILEMAX + asIndex * TILEX + tx));
		}

		for (int bsIndex = 0; bsIndex < (TILEMAX / TILEY); bsIndex++)
		{
		//int bsIndex = 1;

			Bs[TILEY * bsIndex + ty][tx] = mem2d(bd, m, (index * TILEMAX + bsIndex * TILEY + ty), (bx * TILEX + tx));
		}
		
		__syncthreads();
		for (int k = 0; k < TILEMAX; k++)
		{
			temp += As[ty][k] * Bs[k][tx];
		}
		__syncthreads();
	}
	mem2d(cd, m, (by * TILEY + ty), (bx * TILEX + tx)) = temp;
}
